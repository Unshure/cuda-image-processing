#include "hip/hip_runtime.h"
#include "cudaProcess.h"

__global__
void execCudaGrayscale(unsigned char* image, unsigned char* grayImage, int rows, int cols, int channels, int step) {

    int index = threadIdx.x;
    int stride = blockDim.x;

    int numPixels = rows * cols;

    for (int i = index; i < numPixels; i+= stride) {
        int y = index / cols;
        int x = index % cols;

        int blue = (int)image[channels*x + step*y];
        int green = (int)image[channels*x + step*y + 1];
        int red = (int)image[channels*x + step*y + 2];
        
        grayImage[x + cols*y] = (unsigned char)(.3*red) + (.59 * green) + (.11 * blue);
    }
}

__device__
void cudaKernelSum(unsigned char* image, int rows, int cols, int channels, int step, int x, int y, int size, int* sum) {
    int numPixels = 0;
    for (int i = (x - (size/2)); i < (x + (size/2))+1; i++) {
        for (int j = (y - (size/2)); j < (y + (size/2))+1; j++) {
            if (i >= 0 && j >= 0 && i < cols && j < rows) {
                sum[0] += image[i*channels + y*step];
                sum[1] += image[i*channels + y*step + 1];
                sum[2] += image[i*channels + y*step + 2];
                numPixels++;
            }
        }
    }
    sum[0] = sum[0] / numPixels;
    sum[1] = sum[1] / numPixels;
    sum[2] = sum[2] / numPixels;
}

__global__
void execCudaBlur(unsigned char* image, unsigned char* blurImage, int rows, int cols, int channels, int step, int size) {

    int index = threadIdx.x;
    int stride = blockDim.x;

    int numPixels = rows * cols;

    int *sum = (int*)malloc(3 * sizeof(int));
    memset(sum, 0, 3*sizeof(int));

    for (int i = index; i < numPixels; i += stride) {
        int y = index / cols;
        int x = index % cols;

        cudaKernelSum(image, rows, cols, channels, step, x, y, size, sum);
        blurImage[channels*x + step*y] =     sum[0];
        blurImage[channels*x + step*y + 1] = sum[1];
        blurImage[channels*x + step*y + 2] = sum[2];
    }
}

__device__
void cudaKernelLineDetect(unsigned char* image, int rows, int cols, int x, int y, int* val) {
    int sum = 0;
    int numPixels = 0;
    int kx = 0;
    int cudaKernelArray[4][3][3] = {{{-1,-1,-1},{2,2,2},{-1,-1,-1}},
                            {{-1,2,-1},{-1,2,-1},{-1,2,-1}},
                            {{-1,-1,2},{-1,2,-1},{2,-1,-1}},
                            {{2,-1,-1},{-1,2,-1},{-1,-1,2}}};
    for (int i = (x - 1); i < (x + 2); i++) {
        int ky = 0;
        for (int j = (y - 1); j < (y + 2); j++) {
            if (i >= 0 && j >= 0 && i < cols && j < rows) {
                for(int k = 0; k < 4; k ++) {
                    sum += cudaKernelArray[k][kx][ky] * image[i + cols*j];
                }
                numPixels++;
            }
            ky++;
        }
        kx++;
    }
    *val = sum / (numPixels*4);
}

__global__
void execCudaDetectLine(unsigned char* image, unsigned char* lineImage, int rows, int cols, int channels, int step) {
    //Assuming gray image input

    int index = threadIdx.x;
    int stride = blockDim.x;

    int numPixels = rows * cols;
    int *val;

    for(int i = index; i < numPixels; i+= stride) {
        int y = index / cols;
        int x = index % cols;
        cudaKernelLineDetect(image, rows, cols, x, y, val);
        lineImage[x + cols*y] = *val;
    }
}

unsigned char* cudaGrayscale(unsigned char* image, int rows, int cols, int channels, int step) {

    int threadsPerBlock = 1024;
    int numBlocks = 65000;

    unsigned char* cudaImage;
    unsigned char* cudaGrayImage;
    hipMallocManaged(&cudaImage, sizeof(unsigned char)*rows*cols);
    hipMallocManaged(&cudaGrayImage, sizeof(unsigned char)*rows*cols*channels);
    
    hipMemcpy(cudaImage, image, sizeof(unsigned char)*rows*cols*channels, hipMemcpyHostToDevice);
    memset(cudaGrayImage, 0, sizeof(unsigned char)*rows*cols);

    execCudaGrayscale<<<numBlocks, threadsPerBlock>>>(cudaImage, cudaGrayImage, rows, cols, channels, step);
    hipDeviceSynchronize();

    unsigned char* grayImage = (unsigned char*)malloc(sizeof(unsigned char)*rows*cols);
    hipMemcpy(grayImage, cudaGrayImage, sizeof(unsigned char)*rows*cols, hipMemcpyDeviceToHost);

    hipFree(cudaImage);
    hipFree(cudaGrayImage);
    return grayImage;

}

unsigned char* cudaBlur(unsigned char* image, int rows, int cols, int channels, int step, int size) {

    int threadsPerBlock = 1024;
    int numBlocks = 65000;

    unsigned char* cudaImage;
    unsigned char* cudaBlurImage;
    hipMallocManaged(&cudaImage, sizeof(unsigned char)*rows*cols*channels);
    hipMallocManaged(&cudaBlurImage, sizeof(unsigned char)*rows*cols*channels);

    hipMemcpy(cudaImage, image, sizeof(unsigned char)*rows*cols*channels, hipMemcpyHostToDevice);
    memset(cudaBlurImage, 0, sizeof(unsigned char)*rows*cols*channels);

    execCudaBlur<<<numBlocks, threadsPerBlock>>>(cudaImage, cudaBlurImage, rows, cols, channels, step, size);
    hipDeviceSynchronize();

    unsigned char* blurImage = (unsigned char*)malloc(sizeof(unsigned char)*rows*cols*channels);
    hipMemcpy(blurImage, cudaBlurImage, sizeof(unsigned char)*rows*cols*channels, hipMemcpyDeviceToHost);

    hipFree(cudaImage);
    hipFree(cudaBlurImage);
    return blurImage;

}

unsigned char* cudaDetectLine(unsigned char* image, int rows, int cols, int channels, int step) {

    int threadsPerBlock = 1024;
    int numBlocks = 65000;

    unsigned char* grayImage = cudaGrayscale(image, rows, cols, channels, step);

    unsigned char* cudaImage;
    unsigned char* cudaLineImage;
    hipMallocManaged(&cudaImage, sizeof(unsigned char)*rows*cols);
    hipMallocManaged(&cudaLineImage, sizeof(unsigned char)*rows*cols);

    hipMemcpy(cudaImage, grayImage, sizeof(unsigned char)*rows*cols, hipMemcpyHostToDevice);
    memset(cudaLineImage, 0, sizeof(unsigned char)*rows*cols);

    execCudaDetectLine<<<numBlocks, threadsPerBlock>>>(cudaImage, cudaLineImage, rows, cols, channels, step);
    hipDeviceSynchronize();

    unsigned char* lineImage = (unsigned char*)malloc(sizeof(unsigned char)*rows*cols);
    hipMemcpy(lineImage, cudaLineImage, sizeof(unsigned char)*rows*cols, hipMemcpyDeviceToHost);

    hipFree(cudaImage);
    hipFree(cudaLineImage);
    return lineImage;

}
